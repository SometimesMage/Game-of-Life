#include "hip/hip_runtime.h"
extern "C" {
    #include "gpu-frame-computer.h"
}

#include <stdio.h>
#include "hip/hip_runtime.h"

void checkCudaError(hipError_t error);

extern "C"
double gpuComputeFrame(Game *game)
{
    char *d_in, *d_out;

    checkCudaError(hipMalloc(&d_in, (game->width + 2) * (game->height + 2) * sizeof(char)));
    checkCudaError(hipMalloc(&d_out, (game->width + 2) * (game->height + 2) * sizeof(char)));

    checkCudaError(hipMemset(d_out, 0, (game->width + 2) * (game->height + 2) * sizeof(char)));
    checkCudaError(hipMemcpy(d_in, game->data->data, (game->width + 2) * (game->height + 2) * sizeof(char), hipMemcpyHostToDevice));

    dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 grid(ceil(game->width/(float)BLOCK_SIZE), ceil(game->height/(float)BLOCK_SIZE), 1);
    int sharedMemorySize = (BLOCK_SIZE + 2) * (BLOCK_SIZE * 2) * sizeof(char);

    double startTime = currentTime();

    computeFrame2<<<grid, block, sharedMemorySize>>>(d_in, d_out, game->width, game->height);

    double endTime = currentTime();

    checkCudaError(hipGetLastError());

    checkCudaError(hipMemcpy(game->data->data, d_out, (game->width + 2) * (game->height + 2) * sizeof(char), hipMemcpyDeviceToHost));

    checkCudaError(hipFree(d_in));
    checkCudaError(hipFree(d_out));

    return endTime - startTime;
}

void checkCudaError(hipError_t error)
{
    if(error != 0) {
        printf("Cuda Error: %s\n", hipGetErrorName(error));
    }
}