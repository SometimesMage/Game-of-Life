#include "hip/hip_runtime.h"
extern "C" {
    #include "gpu-frame-computer.h"
}

#include "hip/hip_runtime.h"

extern "C"
void gpuComputeFrame(Game *game)
{
    char *d_in, *d_out;

    hipMalloc(&d_in, (game->width + 2) * (game->height + 2) * sizeof(char));
    hipMalloc(&d_out, (game->width + 2) * (game->height + 2) * sizeof(char));

    hipMemset(d_out, 0, (game->width + 2) * (game->height + 2) * sizeof(char));
    hipMemcpy(d_in, game->data->data, (game->width + 2) * (game->height + 2) * sizeof(char), hipMemcpyHostToDevice);

    dim3 block(128, 1, 1);
    dim3 grid(ceil(game->width/(float)128), game->height, 1);

    computeFrame<<<grid, block>>>(d_in, d_out, game->width, game->height);

    hipMemcpy(game->data->data, d_out, (game->width + 2) * (game->height + 2) * sizeof(char), hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);
}