#include "hip/hip_runtime.h"
extern "C" {
    #include "gpu-frame-computer.h"
}

#include <stdio.h>
#include "hip/hip_runtime.h"

void checkCudaError(hipError_t error);

extern "C"
void gpuComputeFrame(Game *game)
{
    char *d_in, *d_out;

    checkCudaError(hipMalloc(&d_in, (game->width + 2) * (game->height + 2) * sizeof(char)));
    checkCudaError(hipMalloc(&d_out, (game->width + 2) * (game->height + 2) * sizeof(char)));

    checkCudaError(hipMemset(d_out, 0, (game->width + 2) * (game->height + 2) * sizeof(char)));
    checkCudaError(hipMemcpy(d_in, game->data->data, (game->width + 2) * (game->height + 2) * sizeof(char), hipMemcpyHostToDevice));

    dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 grid(ceil(game->width/(float)BLOCK_SIZE), ceil(game->height/(float)BLOCK_SIZE), 1);
    int sharedMemorySize = (BLOCK_SIZE + 2) * (BLOCK_SIZE * 2) * sizeof(char);

    computeFrame2<<<grid, block, sharedMemorySize>>>(d_in, d_out, game->width, game->height);
    checkCudaError(hipGetLastError());

    checkCudaError(hipMemcpy(game->data->data, d_out, (game->width + 2) * (game->height + 2) * sizeof(char), hipMemcpyDeviceToHost));

    checkCudaError(hipFree(d_in));
    checkCudaError(hipFree(d_out));
}

void checkCudaError(hipError_t error)
{
    if(error != 0) {
        printf("Cuda Error: %s\n", hipGetErrorName(error));
    }
}