#include "hip/hip_runtime.h"
extern "C" {
    #include "gpu-frame-computer.h"
}

#include <stdio.h>
#include "hip/hip_runtime.h"

void checkCudaError(hipError_t error, const char *file, int line);

extern "C"
double gpuComputeFrame(Game *game)
{
    char *d_in, *d_out;

    checkCudaError(hipMalloc(&d_in, (game->width + 2) * (game->height + 2) * sizeof(char)), __FILE__, __LINE__);
    checkCudaError(hipMalloc(&d_out, (game->width + 2) * (game->height + 2) * sizeof(char)), __FILE__, __LINE__);

    checkCudaError(hipMemset(d_out, 0, (game->width + 2) * (game->height + 2) * sizeof(char)), __FILE__, __LINE__);
    checkCudaError(hipMemcpy(d_in, game->data->data, (game->width + 2) * (game->height + 2) * sizeof(char), hipMemcpyHostToDevice), __FILE__, __LINE__);

    dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 grid(ceil(game->width/(float)BLOCK_SIZE), ceil(game->height/(float)BLOCK_SIZE), 1);
    int sharedMemorySize = (BLOCK_SIZE + 2) * (BLOCK_SIZE * 2) * sizeof(char);

    double startTime = currentTime();

    computeFrame2<<<grid, block, sharedMemorySize>>>(d_in, d_out, game->width, game->height);

    double endTime = currentTime();

    checkCudaError(hipGetLastError(), __FILE__, __LINE__);

    checkCudaError(hipMemcpy(game->data->data, d_out, (game->width + 2) * (game->height + 2) * sizeof(char), hipMemcpyDeviceToHost), __FILE__, __LINE__);

    checkCudaError(hipFree(d_in), __FILE__, __LINE__);
    checkCudaError(hipFree(d_out), __FILE__, __LINE__);

    return endTime - startTime;
}

void checkCudaError(hipError_t error, const char *file, int line)
{
    if(error != 0) {
        printf("Cuda Error: %s (%s:%d)\n", hipGetErrorName(error), file, line);
    }
}